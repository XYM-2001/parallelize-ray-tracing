#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cmath>
#include <chrono>

// Vec3 class for vector operations
struct Vec3 {
    float x, y, z;
    __host__ __device__ Vec3() : x(0), y(0), z(0) {}
    __host__ __device__ Vec3(float x, float y, float z) : x(x), y(y), z(z) {}
    __host__ __device__ Vec3 operator+(const Vec3& b) const { return Vec3(x + b.x, y + b.y, z + b.z); }
    __host__ __device__ Vec3 operator-(const Vec3& b) const { return Vec3(x - b.x, y - b.y, z - b.z); }
    __host__ __device__ Vec3 operator*(float s) const { return Vec3(x * s, y * s, z * s); }
    __host__ __device__ Vec3 operator/(float s) const { return Vec3(x / s, y / s, z / s); }
    __host__ __device__ float dot(const Vec3& b) const { return x * b.x + y * b.y + z * b.z; }
    __host__ __device__ Vec3 normalize() const {
        float mag = sqrt(x * x + y * y + z * z);
        return *this / mag;
    }
};

// Sphere structure
struct Sphere {
    Vec3 center;
    float radius;
    Vec3 color;
    __host__ __device__ bool intersect(const Vec3& origin, const Vec3& dir, float& t) const {
        Vec3 oc = origin - center;
        float a = dir.dot(dir);
        float b = 2.0f * oc.dot(dir);
        float c = oc.dot(oc) - radius * radius;
        float discriminant = b * b - 4 * a * c;
        if (discriminant < 0) return false;
        t = (-b - sqrt(discriminant)) / (2.0f * a);
        return t > 0;
    }
};

// CUDA kernel for rendering
__global__ void render(Vec3* framebuffer, int width, int height, Sphere* spheres, int sphere_count) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * width + x;
    float u = (x + 0.5f) / width;
    float v = (y + 0.5f) / height;
    Vec3 ray_origin(0, 0, 0);
    Vec3 ray_dir = Vec3(u - 0.5f, v - 0.5f, -1).normalize();

    Vec3 pixel_color(0, 0, 0);
    float t_min = 1e20f;

    for (int i = 0; i < sphere_count; ++i) {
        float t;
        if (spheres[i].intersect(ray_origin, ray_dir, t) && t < t_min) {
            t_min = t;
            pixel_color = spheres[i].color;
        }
    }

    framebuffer[idx] = pixel_color;
}

// Host function
void save_image(const Vec3* framebuffer, int width, int height, const char* filename) {
    std::ofstream file(filename, std::ios::out | std::ios::binary);
    file << "P6\n" << width << " " << height << "\n255\n";
    for (int i = 0; i < width * height; ++i) {
        file.put(static_cast<unsigned char>(fminf(framebuffer[i].x, 1.0f) * 255));
        file.put(static_cast<unsigned char>(fminf(framebuffer[i].y, 1.0f) * 255));
        file.put(static_cast<unsigned char>(fminf(framebuffer[i].z, 1.0f) * 255));
    }
    file.close();
}

int main() {
    int width = 7680;
    int height = 4320;
    int num_pixels = width * height;

    auto start = std::chrono::high_resolution_clock::now();

    Vec3* framebuffer;
    hipMallocManaged(&framebuffer, num_pixels * sizeof(Vec3));

    Sphere spheres[] = {
        { Vec3(0, 0, -3), 1, Vec3(1, 0, 0) },
        { Vec3(2, 0, -4), 1, Vec3(0, 1, 0) },
        { Vec3(-2, 0, -4), 1, Vec3(0, 0, 1) }
    };
    Sphere* d_spheres;
    hipMalloc(&d_spheres, sizeof(spheres));
    hipMemcpy(d_spheres, spheres, sizeof(spheres), hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
    render<<<blocks, threads>>>(framebuffer, width, height, d_spheres, 3);

    hipDeviceSynchronize();

    save_image(framebuffer, width, height, "output.ppm");

    hipFree(framebuffer);
    hipFree(d_spheres);

    // 停止计时
    auto end = std::chrono::high_resolution_clock::now();

    // 输出时间
    std::cout << "Total execution time: "
            << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count()
            << " ms\n";
            
    return 0;
}
